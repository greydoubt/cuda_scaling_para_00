#include <iostream>
#include <hip/hip_runtime.h>

__global__ void addOneKernel(int *device_a, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        device_a[idx] += 1;
    }
}

void initializeOnHost(int *host_a, int N) {
    for (int i = 0; i < N; ++i) {
        host_a[i] = i;  // Initialize with some values, e.g., 0, 1, 2, ...
    }
}

void verifyOnHost(int *host_a, int N) {
    for (int i = 0; i < N; ++i) {
        if (host_a[i] != i + 1) {  // Check if each element was incremented by 1
            std::cerr << "Error: host_a[" << i << "] = " << host_a[i] << " (expected " << i + 1 << ")\n";
            return;
        }
    }
    std::cout << "Verification successful! All elements are incremented by 1.\n";
}

int main() {
    const int N = 1024;           // Number of elements
    const size_t size = N * sizeof(int);  // Size of the array in bytes
    int *host_a, *device_a;

    // Allocate memory
    hipMalloc(&device_a, size);         // Device memory
    hipHostMalloc(&host_a, size, hipHostMallocDefault);       // Pinned host memory

    // Initialize host array
    initializeOnHost(host_a, N);

    // Copy data from host to device
    hipMemcpy(device_a, host_a, size, hipMemcpyHostToDevice);

    // Define kernel launch parameters
    int blocks = (N + 255) / 256;   // Calculate number of blocks
    int threads = 256;               // Number of threads per block

    // Launch the kernel to add 1 to each element in device_a
    addOneKernel<<<blocks, threads>>>(device_a, N);

    // Wait for GPU to finish
    hipDeviceSynchronize();

    // Copy the results back to host
    hipMemcpy(host_a, device_a, size, hipMemcpyDeviceToHost);

    // Verify the result
    verifyOnHost(host_a, N);

    // Free allocated memory
    hipFree(device_a);
    hipHostFree(host_a);

    return 0;
}
