#include "hip/hip_runtime.h"
//your function pointer type - returns unsigned char, takes parameters of type unsigned char and float
typedef unsigned char(*pointFunction_t)(unsigned char, float);

//some device function to be pointed to
__device__ unsigned char
Threshold(unsigned char in, float thresh)
{
   ...
}

//pComputeThreshold is a device-side function pointer to your __device__ function
__device__ pointFunction_t pComputeThreshold = Threshold;
//the host-side function pointer to your __device__ function
pointFunction_t h_pointFunction;

//in host code: copy the function pointers to their host equivalent
hipMemcpyFromSymbol(&h_pointFunction, HIP_SYMBOL(pComputeThreshold), sizeof(pointFunction_t))

//You can then pass the h_pointFunction as a parameter to your kernel, which can use it to call your __device__ function.

//your kernel taking your __device__ function pointer as a parameter
__global__ void kernel(pointFunction_t pPointOperation)
{
    unsigned char tmp;
    ...
    tmp = (*pPointOperation)(tmp, 150.0)
    ...
}

//invoke the kernel in host code, passing in your host-side __device__ function pointer
kernel<<<...>>>(h_pointFunction);
