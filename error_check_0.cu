some_kernel<<<(N+255)/256, 256>>>(N, 2.0, d_x, d_y);

hipError_t errSync  = hipGetLastError();
hipError_t errAsync = hipDeviceSynchronize();

if (errSync != hipSuccess) 
  printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
if (errAsync != hipSuccess)
  printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
