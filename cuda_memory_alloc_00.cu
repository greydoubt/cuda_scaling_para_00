// Allocate Unified Memory -- accessible from CPU or GPU
float *x, *y, *sum;
hipMallocManaged(&x, N*sizeof(float));
hipMallocManaged(&y, N*sizeof(float));
 
...
 
// Free memory
hipFree(x);
hipFree(y);
