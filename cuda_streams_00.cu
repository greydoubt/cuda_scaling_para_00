#include <stdio.h>

int main(){
  hipStream_t stream;       // CUDA streams are of type `hipStream_t`.
  hipStreamCreate(&stream); // Note that a pointer must be passed to `cudaCreateStream`.
  someKernel<<<number_of_blocks, threads_per_block, 0, stream>>>(); // `stream` is passed as 4th EC argument.
  hipStreamDestroy(stream); // Note that a value, not a pointer, is passed to `cudaDestroyStream`.
}
