#include "hip/hip_runtime.h"
template <typename P>
struct cudaCallableFunctionPointer
{
public:
  cudaCallableFunctionPointer(P* f_)
  {
    P* host_ptr = (P*)malloc(sizeof(P));
    hipMalloc((void**)&ptr, sizeof(P));

    hipMemcpyFromSymbol(host_ptr, HIP_SYMBOL(*f_), sizeof(P));
    hipMemcpy(ptr, host_ptr, sizeof(P), hipMemcpyHostToDevice);
    
    hipFree(host_ptr)
  }

  ~cudaCallableFunctionPointer()
  {
    hipFree(ptr);
  }

  P* ptr;
};


__device__ double func1(double x)
{
    return x + 1.0f;
}

typedef double (*func)(double x);
__device__ func f_ = func1;

__global__ void test_kernel(func* f)
{
    double x = (*f)(2.0);
    printf("%g\n", x);
}



int main()
{
    cudaCallableFunctionPointer<func> f(&f_);
    test_kernel << < 1, 1 >> > (f.ptr);
}
